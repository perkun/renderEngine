#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
#define CUDA_VERSION 5000
#define GLM_FORCE_RADIANS

#include <stdio.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <time.h>

#include <GL/glew.h>
//#include "/home/grzeslaff/instalki/glfw-3.1.2/include/GLFW/glfw3.h"
#include <GLFW/glfw3.h>
#include<X11/X.h>
#include<X11/Xlib.h>

#include <cuda_gl_interop.h>

#include "RenderEngine.h"
#include "Cuda.h"





int DISPLAY_WIDTH = 512 * 2;
int DISPLAY_HEIGHT = 512 * 2;


	/** ******************** INSTRUKCJA OBSŁUGI: **********************
	 *
	 * 1) stworzyć silnik graficzny (klasa RenderEngine)
	 * 2) stworzyć shader, co najmniej jeden (klasa Shader)
	 * 3) dodać model planetoidy
	 * 4) dodać kamery; pierwsza to obserwator, druga to światło
	 * 5) zlinkować model z sharderem (funkcja linkShaderToModel)
	 * 6) zlinkować kamery z shaderami (funkcja linkCameraToShader)
	 * 7) stworzyć silnik do postprocessingu w CUDA (klasa Cuda)
	 * 8) RenderEngine.renderScene tworzy scene ze wszystkich modeli
	 *
	******************************************************************/
int main() {
	clock_t start_t, end_t;
	double cpu_time_used, cpu_ticks;
	
	
//    int shadow_size = 1024 * 1;

	start_t = clock();
    ///////////////// przygotowanie modelu z pliku bartczakowego //////////////
    int trojkaty[10000][3];
    float punkty[5000][3];
    FILE *asteroid = fopen("../data/model.obj", "r");
    int num_indices, num_vertices;

    fscanf(asteroid, "%d %d", &num_vertices, &num_indices);
//     printf("%d %d\n", num_vertices, num_indices);


    for (int i = 0; i < num_vertices; i++) {
        fscanf(asteroid, "%f %f %f ", &punkty[i][0], &punkty[i][1], &punkty[i][2]);
    }
    for (int i = 0; i < num_indices; i++) {
        fscanf(asteroid, "%d %d %d ", &trojkaty[i][0], &trojkaty[i][1], &trojkaty[i][2]);
    }
    fclose(asteroid);
    ///////////////////////////////////////////////////////////////////////////




//     RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT);
//     RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT, false, true);		// brak wyśw., LICZY jasnosci
    RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT, true, false);		// wyswietla, NIE LICZY jasnosci
    cudaGLSetGLDevice(0);

    int basicShader = render.addShader("basicShader");
    int aster = render.addModel(punkty, num_vertices, trojkaty, num_indices);

    glm::vec3 asteroid_pos(0,2,0);
	asteroid_pos *= 10000.;
	render.models[aster]->transform.setAngles(20, 25,  0, 0);
    render.models[aster]->transform.setPosition(asteroid_pos);

    float d = 1., model_camera_distance, model_light_distance;
	glm::vec3 light_position(0, 0, 0.);
	light_position *= 10000.;
	glm::vec3 camera_position(2, 2, 0);
	camera_position *= 10000.;
	model_camera_distance = glm::length(asteroid_pos - camera_position);
	model_light_distance = glm::length(asteroid_pos - light_position);
// 	render.addCamera(camera_position, asteroid_pos, 70.0f, (float)DISPLAY_WIDTH/(float)DISPLAY_HEIGHT, model_light_distance - 5, model_light_distance + 5);
	render.addCamera(camera_position, asteroid_pos, -d, d, -d, d, model_camera_distance - 1, model_camera_distance + 1);
	render.addCamera(light_position, asteroid_pos, -d, d, -d, d, model_light_distance - 1, model_light_distance + 1);
    render.linkShaderToModel(basicShader, aster);

    for (int i = 0; i < render.shaders.size(); i++) {
        render.linkCameraToShader(0, i);
        render.linkCameraToShader(1, i);
    }
///////////////////////////////// CUDA /////////////////////////////////////////
	Cuda cuda(render.textures[ render.fb_tx ]->color_texture, DISPLAY_WIDTH, DISPLAY_HEIGHT);
/////////////////////// RENDEROWANIE SCENY /////////////////////////////////////

	int ILOSC_SYMULACJI = 90;
	double jasnosci[30][360];
    int counter = 0;
//     render.render_off_screen = false;
//    while (!render.display.is_closed) {
     for (int k = 0; k < 90; k++) {
 		render.models[aster]->transform.gamma += 2.*M_PI/ILOSC_SYMULACJI;
        render.renderScene();
// 		camera_position.z += (float) k/100;
		model_camera_distance = glm::length(asteroid_pos - camera_position);
		model_light_distance = glm::length(asteroid_pos - light_position);
		render.updateCamera(camera_position, asteroid_pos, -d, d, -d, d, model_camera_distance - 1, model_camera_distance + 1 , 0);
//         if ( counter == 0 ) 
// 		printf("%f", (float)k/360.);
//            dumpToImage();
//  		printf("%f\t%f\n", (float)k/360., cuda.calculateMagnitudes());
		if (render.render_off_screen)
			jasnosci[0][k] = cuda.calculateMagnitudes();
		getchar();
        counter++;

    }
////////////////////////////////////////////////////////////////////////////////////////////////    


// 	for (int i = 0; i < ILOSC_SYMULACJI; i++) 
// 		printf("%d\t%lf\n", i, jasnosci[0][i]);


    end_t = clock();


	cpu_ticks = (double) (end_t - start_t);
	cpu_time_used = ((double) (end_t - start_t)) / CLOCKS_PER_SEC;
//	printf("computation time: %lf\n", cpu_time_used);

// 	for (int k = 0; k < 360; k++)
// 		printf("%.16lf\t", jasnosci[0][k]);
// 	puts("  ");
	
	
	return 0;
}




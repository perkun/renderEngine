#include "hip/hip_runtime.h"
#include "Cuda.h"

#define GLM_FORCE_RADIANS

texture<uchar4, 2, hipReadModeNormalizedFloat> tex_img;

__global__ void kernel(float *result, float *result2, float *result3) {
	int x, y, width = 1000, height = 1000;
	*result = -7.;
	*result2 = -8.;
	*result3 = -22.;
	printf("res3: %f\n", *result3);
	printf("res2: %f\n", *result2);
	
	*result3 = 0;
	for ( x = 0; x < width; x++)
		for ( y = 0; y < height; y++)
			*result3 += tex2D(tex_img, x + 0.5f, y + 0.5f).x;
		
		printf("res3_2: %f\n", *result3);
	
}

__global__ void sumA(float *blocks_partials, int width, int height) {
    //     int dim = blockDim.x;
    __shared__ float partials[512];
    int texture_x, texture_y;
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int local_id = threadIdx.x;
    int i;

    if (global_id < width * height) {

        texture_x = global_id%height;
        texture_y = (int) (global_id/height);

// 	if (local_id == 0)
// 		blocks_partials[blockIdx.x] = 0;

        // kopia textury lokalnie dla bloku (shared mem)
        partials[local_id] = tex2D(tex_img, texture_x + 0.5f, texture_y + 0.5f).x;

        for (i = 1; i <= blockDim.x/2; i *= 2)
            if (local_id%(i*2) == 0) {
                partials[local_id] += partials[local_id + i];
                __syncthreads();
            }

        if (local_id == 0)
            blocks_partials[blockIdx.x] = partials[0];
    }
}

__global__ void sumB(float *blocks_partials, float *suma, int size) {
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
	int local_id = threadIdx.x;
	int i;
	
	if (local_id < 128)	{
	
		
	__shared__ float partials[1024];
	partials[local_id] = blocks_partials[global_id];
	
	
	// 	if (id < size) {
	for (i = 1; i <= blockDim.x/2; i *= 2)
		if (local_id%(i*2) == 0) {
			partials[local_id] += partials[local_id + i];
			__syncthreads();
		}
		// 	}
		
		if (local_id == 0)
			blocks_partials[blockIdx.x] = partials[0];
		
		// 	if (global_id == 0)
		// 		*suma = blocks_partials[0];
	}
}


Cuda::Cuda(GLuint &tex, int w, int h) {
	width = w;
	height = h;
	texture = tex;
	
	hipMalloc( (void**)&dev_result, sizeof(float) );
	
// 	glBindFramebuffer(GL_FRAMEBUFFER, 0);
// 	hipError_t err = hipGraphicsGLRegisterImage(&vbo_res, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
// 	if ( err == hipSuccess)
// 		printf("image successfully registered\n");
// 	else if ( err == hipErrorInvalidHandle)
// 		printf("error registering image, Invalid Handle (duplicate entry)\n");
// 	else if (err == hipErrorUnknown)
// 		printf("error registering image (maped for access)\n");
	
	channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	hipMallocArray( &array_img, &channelDesc, w, h );

	
	threads_per_block = 128;//(DISPLAY_WIDTH / 128) + 1;	// musi być 128*natural_number
	initial_num_blocks =  ceil(   (float)(w*h) / (float)threads_per_block   ); 				// musi być 2^N
// 	printf("num blocks: %d\n", init_num_blocks);
	
	hipMalloc( (void**)&dev_blocks_partials, sizeof(float)* initial_num_blocks );
	
	
	tex_img.addressMode[0] = hipAddressModeBorder;
	tex_img.addressMode[1] = hipAddressModeBorder;
	tex_img.filterMode = hipFilterModeLinear;
	tex_img.normalized = false; // don't access with normalized texture coords
	
	
// 	hipGraphicsMapResources(1, &vbo_res, 0);
// 	hipGraphicsSubResourceGetMappedArray(&array_img, vbo_res, 0, 0);
// 	hipBindTextureToArray(tex_img, (hipArray *)array_img, channelDesc);
// 	if (hipGraphicsMapResources(1, &vbo_res, 0) != hipSuccess) printf("error maping resources\n");
// 	err = hipGraphicsMapResources(1, &vbo_res, 0);
// 	if ( err == hipSuccess)
// 		printf("image successfully maped\n");
// 	else if ( err == hipErrorInvalidHandle)
// 		printf("error maping image, Invalid Handle (duplicate entry)\n");
// 	else if (err == hipErrorUnknown)
// 		printf("error maping image (maped for access)\n");

}

float Cuda::calculateMagnitudes() {
// 	hipMalloc( (void**)&dev_blocks_partials, sizeof(float)* initial_num_blocks );
// 	hipMallocArray( &array_img, &channelDesc, width, height );
	
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	hipError_t err = hipGraphicsGLRegisterImage(&vbo_res, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
	if ( err == hipErrorInvalidHandle)
		printf("error registering image, Invalid Handle (duplicate entry)\n");
	else if (err == hipErrorUnknown)
		printf("error registering image (maped for access)\n");
	
// 	hipMallocArray( &array_img, &channelDesc, width, height );
	
// 	if (hipGraphicsMapResources(1, &vbo_res, 0) != hipSuccess) printf("error maping resources\n");
	err = hipGraphicsMapResources(1, &vbo_res, 0);
	if ( err == hipErrorInvalidHandle)
		printf("error maping image, Invalid Handle (duplicate entry)\n");
	else if (err == hipErrorUnknown)
		printf("error maping image (maped for access)\n");
	
	if (hipGraphicsSubResourceGetMappedArray(&array_img, vbo_res, 0, 0) != hipSuccess) printf("error getting mapped array\n");
	if (hipBindTextureToArray(tex_img, (hipArray *)array_img, channelDesc) != hipSuccess) printf("error binding to array\n");
	
	
	num_blocks =  initial_num_blocks; 
	sumA<<< num_blocks, threads_per_block >>>(dev_blocks_partials, width, height);
	hipDeviceSynchronize();
	while (num_blocks > 1) {
		num_blocks = ceil((float)num_blocks/(float)threads_per_block);
// 		printf("number of blocks: %d\n", num_blocks);
		sumB<<< num_blocks, threads_per_block >>>(dev_blocks_partials, dev_result, num_blocks);
		hipDeviceSynchronize();
	}
	
	
	copyResult();
	
	hipUnbindTexture(tex_img);
	hipGraphicsUnmapResources(1, &vbo_res, 0);
	hipGraphicsUnregisterResource(vbo_res);
	
// 	hipFreeArray(array_img);
// 	hipFree(dev_blocks_partials);
	
	return -2.5*log10(result);
}


void Cuda::copyResult() {
	hipMemcpy(&result, &dev_blocks_partials[0], sizeof(float), hipMemcpyDeviceToHost);
// 	printf("sum of block partials (DEVICE): %f\n", result);
}


Cuda::~Cuda() {
	hipUnbindTexture(tex_img);
	hipFree(dev_blocks_partials);
	hipFree(dev_result);
	//hipFreeArray(array_img);
}
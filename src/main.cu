#include "hip/hip_runtime.h"
/** \file */
/** ******************** INSTRUKCJA OBSŁUGI: **********************
 *
 * 1. stworzyć silnik graficzny (klasa RenderEngine)
 * 2. stworzyć shader, co najmniej jeden (klasa Shader)
 * 3. dodać model planetoidy
 * 4. dodać kamery; pierwsza to obserwator, druga to światło
 * 5. zlinkować model z sharderem (funkcja linkShaderToModel)
 * 6. zlinkować kamery z shaderami (funkcja linkCameraToShader)
 * 7. stworzyć silnik do postprocessingu w CUDA (klasa Cuda)
 * 8. RenderEngine.renderScene tworzy scene ze wszystkich modeli
 *
******************************************************************/

// #include <hip/hip_runtime.h>
#define CUDA_VERSION 5000
#define GLM_FORCE_RADIANS

#include <stdio.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <time.h>

#include <unistd.h>

#include <GL/glew.h>
//#include "/home/grzeslaff/instalki/glfw-3.1.2/include/GLFW/glfw3.h"
#include <GLFW/glfw3.h>
#include<X11/X.h>
#include<X11/Xlib.h>

#include <cuda_gl_interop.h>

#include "RenderEngine.h"
#include "Cuda.h"


int DISPLAY_WIDTH = 512 * 2;
int DISPLAY_HEIGHT = 512 * 2;

bool Mesh::normalize = true;
bool Mesh::normal_interpolation = true;

int main() {
	int k;
	clock_t start_t, end_t;
    double cpu_time_used, cpu_ticks;


//    int shadow_size = 1024 * 1;

    start_t = clock();



//     RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT, false, true);		// brak wyśw., LICZY jasnosci
//     RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT, true, false);		// wyswietla, NIE LICZY jasnosci
	RenderEngine render(DISPLAY_WIDTH, DISPLAY_HEIGHT, true, false, true);
	cudaGLSetGLDevice(0);

    int basicShader = render.addShader("basicShader");
//     int aster = render.addModel(punkty, num_vertices, trojkaty, num_indices);
    int aster = render.addModelAsteroidFormat("../data/hw1_3.obj");


    glm::vec3 asteroid_pos(0,2,0);
    asteroid_pos *= 10000.;
    render.models[aster]->transform.setAngles(20, 25,  0, 0);
    render.models[aster]->transform.setPosition(asteroid_pos);

    float d = 1., model_camera_distance, model_light_distance;
    glm::vec3 light_position(2., 0., 0.);
    light_position *= 10000.;
    glm::vec3 camera_position(0., 0., 0.);
    camera_position *= 10000.;
    model_camera_distance = glm::length(asteroid_pos - camera_position);
    model_light_distance = glm::length(asteroid_pos - light_position);
// // 	render.addCamera(camera_position, asteroid_pos, 70.0f, (float)DISPLAY_WIDTH/(float)DISPLAY_HEIGHT, model_light_distance - 5, model_light_distance + 5);
    render.addCamera(camera_position, asteroid_pos, -d, d, -d, d,
						model_camera_distance - 1, model_camera_distance + 1);
    render.addCamera(light_position, asteroid_pos, -d, d, -d, d,
						model_light_distance - 1, model_light_distance + 1);
    render.linkShaderToModel(basicShader, aster);

    render.linkBasicCamerasToShader();




	/////////////////////////////////

// 	render.textures[render.shadow_texture_id]->setUniform(
// 			*render.shaders[basicShader], "color_shadow", "depth");


// 	render.addTexture(100, 100);


    ///////////// CUDA ////////////////////////
    Cuda cuda(render.textures[ render.fb_tx ]->color_texture, DISPLAY_WIDTH, DISPLAY_HEIGHT);


    /////////////////////// RENDEROWANIE SCENY /////////////////////////

    int ILOSC_SYMULACJI = 90;

    double jasnosci[30][360];
    int counter = 0;
//     render.render_off_screen = false;
//    while (!render.display.is_closed) {

	while (!render.display.is_closed)
	{
//     for (int k = 0; k < 2; k++) {
//      for (int k = 0; k < 90; k++) {
//         render.models[aster]->transform.gamma += 2.*M_PI/ILOSC_SYMULACJI;
        render.renderScene();

        model_camera_distance = glm::length(asteroid_pos - camera_position);
        model_light_distance = glm::length(asteroid_pos - light_position);
//         render.updateCamera(camera_position, asteroid_pos,
// 							-d, d, -d, d, model_camera_distance - 1,
// 							model_camera_distance + 1 , 0);


         if (render.render_off_screen)
             jasnosci[0][k] = cuda.calculateMagnitudes();

// 		sleep(1);
        counter++;
    }


    if (render.render_off_screen)
	{
		for (int i = 0; i < ILOSC_SYMULACJI; i++)
			printf("%d\t%lf\n", i, jasnosci[0][i]);
	}





    end_t = clock();


    cpu_ticks = (double) (end_t - start_t);
    cpu_time_used = ((double) (end_t - start_t)) / CLOCKS_PER_SEC;
//	printf("computation time: %lf\n", cpu_time_used);

// 	for (int k = 0; k < 360; k++)
// 		printf("%.16lf\t", jasnosci[0][k]);
// 	puts("  ");


    return 0;
}



